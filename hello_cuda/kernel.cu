#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include "sphere.h"
#include "box.h"
#include "3d_heart.h"
#include "camera.h"
#include "hittable_list.h"
#include "material.h"

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
      << file << ":" << line << " '" << func << "' \n";
    hipDeviceReset();
    exit(99);
  }
}

__global__ void rand_init(hiprandState* rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init(1984, 0, 0, rand_state);
  }
}

__device__ inline float clamp(float x, float min, float max)
{
  if (x < min) return min;
  else if (x > max) return max;
  return x;
}

__device__ vec3 color(const ray& r, hittable** world, hiprandState* local_rand_state)
{
  ray cur_ray = r;
  vec3 emitted(0, 0, 0);
  vec3 cur_attenuation(1, 1, 1);
  for (int i = 0; i < 50; i++) {
    hit_record rec;
    if (!(*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) return vec3(0.0, 0.0, 0.0);
    
    ray scattered;
    vec3 attenuation;
    float pdf;
    emitted += cur_attenuation * rec.mat_ptr->scattering_pdf(cur_ray, rec, scattered)
      * rec.mat_ptr->emitted(rec.u, rec.v, rec.p);

    if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, pdf, local_rand_state)) {
      cur_attenuation *= attenuation;
      cur_ray = scattered;
    }
    else {
      return emitted / pdf;
    }

    //vec3 unit_direction = unit_vector(cur_ray.direction());
    //float t = 0.5f * (unit_direction.y() + 1.0f);
    //vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    //return cur_attenuation * c;
  }
  return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j * max_x + i;
  //Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, const int max_x, const int max_y,
  int ns, camera** cam, hittable** world, hiprandState* rand_state)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j * max_x + i;
  hiprandState local_rand_state = rand_state[pixel_index];
  vec3 col(0, 0, 0);
  for (int s = 0; s < ns; s++) {
    float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x - 1);
    float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y - 1);
    ray r = (*cam)->get_ray(u, v, &local_rand_state);
    col += color(r, world, &local_rand_state);
  }
  rand_state[pixel_index] = local_rand_state;
  col /= float(ns);
  col[0] = clamp(sqrt(col[0]), 0, 0.999);
  col[1] = clamp(sqrt(col[1]), 0, 0.999);
  col[2] = clamp(sqrt(col[2]), 0, 0.999);
  fb[pixel_index] = col;
}

__global__ void create_world(
  hittable** d_list,
  hittable** d_world,
  camera** d_camera,
  int nx, int ny,
  vec3 lookfrom,
  vec3 lookat,
  hiprandState* rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprandState local_rand_state = *rand_state;
    int i = 0;
    d_list[i++] = new yz_rect(0, 555, 0, 555, 555, new lambertian(vec3(.12, .45, .15)));
    d_list[i++] = new yz_rect(0, 555, 0, 555, 0, new lambertian(vec3(.65, .05, .05)));
    d_list[i++] = new xz_rect(113, 443, 127, 432, 554, new diffuse_light(vec3(4, 4, 4)));
    d_list[i++] = new xz_rect(0, 555, 0, 555, 555, new lambertian(vec3(.73, .73, .73)));
    d_list[i++] = new xz_rect(0, 555, 0, 555, 0, new lambertian(vec3(.73, .73, .73)));
    d_list[i++] = new xy_rect(0, 555, 0, 555, 555, new lambertian(vec3(.73, .73, .73)));
    d_list[i++] = new sphere(vec3(271.4, 271.4, 0), 50, new lambertian(vec3(0.8, 0.5, 0.1)));
    //d_list[i++] = new sphere(vec3(90, 66, 21), 25, new lambertian(vec3(0.1, 0.5, 0.8)));

    *rand_state = local_rand_state;
    *d_world = new hittable_list(d_list, i);

    float dist_to_focus = (lookfrom - lookat).length();
    *d_camera = new camera(
      lookfrom,
      lookat,
      vec3(0, 1, 0),
      40,
      float(nx) / float(ny),
      0.1f,
      dist_to_focus);
  }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
  int i = 0;
  delete((yz_rect*)d_list[i])->mat_ptr;
  delete d_list[i++];
  delete((yz_rect*)d_list[i])->mat_ptr;
  delete d_list[i++];
  
  delete((xz_rect*)d_list[i])->mat_ptr;
  delete d_list[i++];
  delete((xz_rect*)d_list[i])->mat_ptr;
  delete d_list[i++];
  delete((xz_rect*)d_list[i])->mat_ptr;
  delete d_list[i++];

  delete((xy_rect*)d_list[i])->mat_ptr;
  delete d_list[i++];
  
  delete((sphere*)d_list[i])->mat_ptr;
  delete d_list[i++];
  //delete((sphere*)d_list[i])->mat_ptr;
  //delete d_list[i++];

  delete* d_world;
  delete* d_camera;
}

int main()
{
  int nx = 800,
    ny = 800,
    ns = 100,
    tx = 8,
    ty = 8;

  int pixels = nx * ny;
  size_t fb_size = pixels * sizeof(vec3);

  vec3* fb;
  checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

  hiprandState* d_rand_state;
  checkCudaErrors(hipMalloc((void**)&d_rand_state, pixels * sizeof(hiprandState)));
  hiprandState* d_rand_state2;
  checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));
  // we need that 2nd random state to be initialized for the world creation
  rand_init<<<1, 1>>>(d_rand_state2);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // make our world of hittables & the camera
  hittable** d_list;
  int num_hittables = 7;
  checkCudaErrors(hipMalloc((void**)&d_list, num_hittables * sizeof(hittable*)));
  hittable** d_world;
  checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));
  camera** d_camera;
  checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
  create_world<<<1, 1>>>(d_list, d_world, d_camera, nx, ny,
    vec3(278, 278, -800), vec3(278, 278, 0), d_rand_state2);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  clock_t start, stop;
  start = clock();

  dim3 blocks(nx / tx + 1, ny / ty + 1);
  dim3 threads(tx, ty);
  render_init <<<blocks, threads>>> (nx, ny, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  render <<<blocks, threads>>> (fb, nx, ny, ns,
    d_camera,
    d_world,
    d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << "took " << timer_seconds << " seconds.\n";

  FILE* fp = fopen("output.ppm", "wb");
  fprintf(fp, "P6 %d %d 255 ", nx, ny);
  for (int j = ny - 1; j >= 0; j--)
    for (int i = 0; i < nx; i++) {
      size_t pixel_index = j * nx + i;
      unsigned char r = int(256 * fb[pixel_index].r());
      unsigned char g = int(256 * fb[pixel_index].g());
      unsigned char b = int(256 * fb[pixel_index].b());
      fprintf(fp, "%c%c%c", r, g, b);
    }
  fclose(fp);

  checkCudaErrors(hipDeviceSynchronize());
  free_world <<<1, 1>>> (d_list, d_world, d_camera);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_camera));
  checkCudaErrors(hipFree(d_rand_state));
  checkCudaErrors(hipFree(d_rand_state2));
  checkCudaErrors(hipFree(fb));
  return 0;
}